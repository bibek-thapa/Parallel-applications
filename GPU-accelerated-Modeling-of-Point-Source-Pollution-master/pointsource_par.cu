#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>


#define THREADS_PER_BLOCK 1024
#define TIME 3600000


__global__ void initialize(float *a_d, float *b_d, float *c_d, int arraySize)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;	
	if(ix==0)
	{
		a_d[ix]=200.0;
		b_d[ix]=200.0;
		
	}

	else if (ix<arraySize)
	{
		a_d[ix]=0.0;
		b_d[ix]=0.0;
	}

}
 

__global__ void compute(float *a_d, float *b_d, float *c_d, int arraySize)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;	
	float temp;
	if( ix > 0 && ix < arraySize-1){
	    temp = (a_d[ix+1]+a_d[ix-1])/2.0;
	    __syncthreads();
	    b_d[ix]=temp;
	    __syncthreads();
	}


				 	
}		




extern "C" void pointsource_pollution (float *a, float *b, int *c, int arraySize)
{

	float *a_d, *b_d, *c_d;

	hipMalloc ((void**) &a_d, sizeof(float) * arraySize);
	hipMalloc ((void**) &b_d, sizeof(float) * arraySize);
	hipMalloc ((void**) &c_d, sizeof(float) * arraySize);
	

	initialize<<< ceil((float) arraySize/THREADS_PER_BLOCK), THREADS_PER_BLOCK >>> (a_d, b_d, c_d, arraySize);

	for(int i=0;i<TIME;i++){

	compute <<< ceil((float) arraySize/THREADS_PER_BLOCK), THREADS_PER_BLOCK >>> (a_d, b_d, c_d, arraySize);
	a_d=b_d;
	
	}

	hipMemcpy (a, a_d, sizeof(float) * arraySize, hipMemcpyDeviceToHost);
	
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf ("CUDA error: %s\n", hipGetErrorString(err));
		
	
	hipFree (a_d);
	hipFree (b_d);
	hipFree (c_d);
		
	
}