// VDotProduct.cu
//
// driver and kernel call

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define THREADS_PER_BLOCK 32

 
__global__ void dot_product (int *a_d, int *b_d, int *c_d, int arraySize)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int t = threadIdx.x;
    __shared__ int temp[THREADS_PER_BLOCK];
  

    if (x < arraySize/2)
	{
	 	a_d[x] = x + 1;
	}
    else 
       {
		a_d[x] = x -((x - arraySize/2)*2);
	
	}
  
  	
		b_d[x] = (x % 10) + 1;

	

	temp[t] = a_d[x] * b_d[x];
	__syncthreads();

	if( threadIdx.x==0)
	{

	int sum=0;
	for(int i=0;i<THREADS_PER_BLOCK;i++)
		{
		sum+=temp[i];

		}
		atomicAdd(c_d,sum);
	}


}

extern "C" void gpu_dot_product (int *a, int *b, int *c, int arraySize)
{
	int *a_d, *b_d, *c_d;

	hipMalloc ((void**) &a_d, sizeof(int) * arraySize);
	hipMalloc ((void**) &b_d, sizeof(int) * arraySize);
	hipMalloc ((void**) &c_d, sizeof(int) * arraySize);
	hipMemcpy (a_d, a, sizeof(int) * arraySize, hipMemcpyHostToDevice);
	hipMemcpy (b_d, b, sizeof(int) * arraySize, hipMemcpyHostToDevice);

	dot_product <<< ceil((float) arraySize/THREADS_PER_BLOCK), THREADS_PER_BLOCK >>> (a_d, b_d, c_d, arraySize);
	
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf ("CUDA error: %s\n", hipGetErrorString(err));
		
	hipMemcpy (c, c_d, sizeof(int) * arraySize, hipMemcpyDeviceToHost);
	hipFree (a_d);
	hipFree (b_d);
	hipFree (c_d);
		
	
}

